#include "hip/hip_runtime.h"

#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8


#include "cutil_math.h"

texture<uchar4, 2, hipReadModeElementType> tex;

int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// 24-bit multiplication is faster on G80,
// but we must be sure to multiply integers
// only within [-8M, 8M - 1] range
#if defined(CUDA_24BIT_IMUL)
#	define IMUL(a, b) __mul24(a, b)
#else
#	define IMUL(a, b) (a)*(b)
#endif

// TODO deformation
// At the moment this function is only used for testing
__global__
void warpImage( uchar4 *d_dst, int imageW, int imageH, float matrix[9] )
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if( ix < imageW && iy < imageH )
    {

        // Warp is only used for TESTING purposes
        unsigned int pos = ix + iy*imageW;
        
        float tu = ix+matrix[0];//float(ix)*cos(0.1) - float(iy)*sin(0.1) +10;
        float tv = iy+matrix[1];//float(ix)*sin(0.1) + float(iy)*cos(0.1);
        uchar4 col =  tex2D( tex, tu, tv ); 
        const float R = (float)col.x*matrix[2];
        const float G = (float)col.y*matrix[2];
        const float B = (float)col.z*matrix[2];
        d_dst[ pos ].x = (unsigned char)( R > 0 ) ? ( ( R <=255 ) ? R : 255 ): 0 ;
        d_dst[ pos ].y = (unsigned char)( G > 0 ) ? ( ( G <=255 ) ? G : 255 ): 0 ;
        d_dst[ pos ].z = (unsigned char)( B > 0 ) ? ( ( B <=255 ) ? B : 255 ): 0 ;
    }
}

__global__
void diffRGB( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        unsigned int posPix1 = ix + iy*imageW;

        const float3 src1 =  make_float3( d_src1[posPix1].x, d_src1[posPix1].y, d_src1[posPix1].z );
        const float3 src2 =  make_float3( d_src2[posPix1].x, d_src2[posPix1].y, d_src2[posPix1].z );

        const float3 result = src1-src2;

        const float R = (result.x+128.f);
        const float G = (result.y+128.f);
        const float B = (result.z+128.f);

        d_dst[ posPix1 ].x = (unsigned char)( R > 0 ) ? ( ( R <=255 ) ? R : 255 ): 0 ;
        d_dst[ posPix1 ].y = (unsigned char)( G > 0 ) ? ( ( G <=255 ) ? G : 255 ): 0 ;
        d_dst[ posPix1 ].z = (unsigned char)( B > 0 ) ? ( ( B <=255 ) ? B : 255 ): 0 ;
    }
}

#include <stdio.h>
// Convert 422 ycbycr to rgb
__global__
void
YCbYCrToRGBA(
    uchar4 *dst,
    uchar4 *src,
    int imageW, //960
    int imageH  //1080
)
{
    // Position in src image (960,1080)
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const float Y1 = (float)src[imageW * iy + ix].x;
        const float Cb = (float)src[imageW * iy + ix].y;
        const float Y2 = (float)src[imageW * iy + ix].z;
        const float Cr = (float)src[imageW * iy + ix].w;

        // Conversion in RGB
        const float R1 = Y1 + 1.371*(Cr-128);
        const float G1 = Y1 - 0.698*(Cr-128) - 0.336*(Cb - 128);
        const float B1 = Y1 + 1.732*(Cb-128);

        const float R2 = Y2 + 1.371*(Cr-128);
        const float G2 = Y2 - 0.698*(Cr-128) - 0.336*(Cb - 128);
        const float B2 = Y2 + 1.732*(Cb-128);

        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        dst[ posPix1 ].x = (unsigned char)( R1 > 0 ) ? ( ( R1 <=255 ) ? R1 : 255 ): 0 ;
        dst[ posPix1 ].y = (unsigned char)( G1 > 0 ) ? ( ( G1 <=255 ) ? G1 : 255 ): 0 ;
        dst[ posPix1 ].z = (unsigned char)( B1 > 0 ) ? ( ( B1 <=255 ) ? B1 : 255 ): 0 ;
        dst[ posPix1 ].w = 0;

        dst[ posPix2 ].x = (unsigned char)( R2 > 0 ) ? ( ( R2 <=255 ) ? R2 : 255 ): 0 ;
        dst[ posPix2 ].y = (unsigned char)( G2 > 0 ) ? ( ( G2 <=255 ) ? G2 : 255 ): 0 ;
        dst[ posPix2 ].z = (unsigned char)( B2 > 0 ) ? ( ( B2 <=255 ) ? B2 : 255 ): 0 ;
        dst[ posPix2 ].w = 0;
    }
}

__global__
void
YCbYCrToY(
    uchar4 *dst,
    uchar4 *src,
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const unsigned char Y1 = src[imageW * iy + ix].x;
        const unsigned char Y2 = src[imageW * iy + ix].z;

        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        dst[ posPix1 ].x = Y1;
        dst[ posPix1 ].y = Y1;
        dst[ posPix1 ].z = Y1;
        dst[ posPix1 ].w = 0;

        dst[ posPix2 ].x = Y2;
        dst[ posPix2 ].y = Y2;
        dst[ posPix2 ].z = Y2;
        dst[ posPix2 ].w = 0;
    }
}


__global__
void
YToYCbYCr(
    uchar4 *dst,
    uchar4 *src,
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        const unsigned char Y1 = src[ posPix1 ].x;
        const unsigned char Y2 = src[ posPix2 ].x;

        dst[imageW * iy + ix].x = Y1;
        dst[imageW * iy + ix].y = 0; // Cb
        dst[imageW * iy + ix].z = Y2;
        dst[imageW * iy + ix].w = 0; // Cr
    }
}

__global__
void
Gray1ToRGBA(
    uchar4 *dst,            // RGB
    unsigned char *src,     // Gray
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const unsigned int position = imageW * iy + ix;
        const unsigned char color = src[position];

        dst[ position ].x = color;
        dst[ position ].y = color;
        dst[ position ].z = color;
        dst[ position ].w = 0;
    }
}

__global__
void
fromDiff(
    uchar4 *dst,
    uchar4 *d_srcA,
    uchar4 *d_srcB,
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const float Y1A = (float)d_srcA[imageW * iy + ix].x;
        const float CbA = (float)d_srcA[imageW * iy + ix].y;
        const float Y2A = (float)d_srcA[imageW * iy + ix].z;
        const float CrA = (float)d_srcA[imageW * iy + ix].w;

        // Conversion in RGB
        const float R1A = Y1A + 1.371*(CrA-128);
        const float G1A = Y1A - 0.698*(CrA-128) - 0.336*(CbA - 128);
        const float B1A = Y1A + 1.732*(CbA-128);

        const float R2A = Y2A + 1.371*(CrA-128);
        const float G2A = Y2A - 0.698*(CrA-128) - 0.336*(CbA - 128);
        const float B2A = Y2A + 1.732*(CbA-128);

        // SRC B
        const float Y1B = (float)d_srcB[imageW * iy + ix].x;
        const float CbB = (float)d_srcB[imageW * iy + ix].y;
        const float Y2B = (float)d_srcB[imageW * iy + ix].z;
        const float CrB = (float)d_srcB[imageW * iy + ix].w;

        // Conversion in RGB
        const float R1B = Y1B + 1.371*(CrB-128);
        const float G1B = Y1B - 0.698*(CrB-128) - 0.336*(CbB - 128);
        const float B1B = Y1B + 1.732*(CbB-128);

        const float R2B = Y2B + 1.371*(CrB-128);
        const float G2B = Y2B - 0.698*(CrB-128) - 0.336*(CbB - 128);
        const float B2B = Y2B + 1.732*(CbB-128);

        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        const float R1 = (R1B-R1A)*0.5 + 127.f;
        const float G1 = (G1B-G1A)*0.5 + 127.f;
        const float B1 = (B1B-B1A)*0.5 + 127.f;
        const float R2 = (R2B-R2A)*0.5 + 127.f;
        const float G2 = (G2B-G2A)*0.5 + 127.f;
        const float B2 = (B2B-B2A)*0.5 + 127.f;

        dst[ posPix1 ].x = (unsigned char)( R1 > 0 ) ? ( ( R1 <=255 ) ? R1 : 255 ): 0 ;
        dst[ posPix1 ].y = (unsigned char)( G1 > 0 ) ? ( ( G1 <=255 ) ? G1 : 255 ): 0 ;
        dst[ posPix1 ].z = (unsigned char)( B1 > 0 ) ? ( ( B1 <=255 ) ? B1 : 255 ): 0 ;

        dst[ posPix2 ].x = (unsigned char)( R2 > 0 ) ? ( ( R2 <=255 ) ? R2 : 255 ): 0 ;
        dst[ posPix2 ].y = (unsigned char)( G2 > 0 ) ? ( ( G2 <=255 ) ? G2 : 255 ): 0 ;
        dst[ posPix2 ].z = (unsigned char)( B2 > 0 ) ? ( ( B2 <=255 ) ? B2 : 255 ): 0 ;
    }
}

__global__
void
RGBAtoFloat(
    float *dst,
    uchar4 *d_src,
    int imageW,
    int imageH,
    int pitch
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH )
    {
        const unsigned int position = imageW * iy + ix;

        const float col = d_src[position].x;
        dst[position] = col/255.0;
    }
}

__global__
void
FloatToRGBA(
    uchar4 *dst,
    float *d_src,
    int imageW,
    int imageH,
    int pitch
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH )
    {
        const unsigned int position = pitch * iy + ix;

        const float pixVal = d_src[position]*255.0;
        const unsigned char p = (unsigned char)( pixVal > 0 ) ? ( ( pixVal <=255 ) ? pixVal : 255 ): 0 ;

        dst[position].x = p;
        dst[position].y = p;
        dst[position].z = p;
        dst[position].w = 0;
    }
}


// Testing purpose
__global__
void
Integrate(
    float *dst,
    float *d_src,
    int imageW,
    int imageH,
    int pitch
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if(ix < imageW && iy < imageH )
    {
        const unsigned int position = pitch * iy + ix;

        float sum = 0;

        for( int i=0; i < ix; i++ )
            sum += d_src[i];
        
        dst[position] = sum;
    }
}


// Transpose of matrix
__global__ void
transpose(
	float *g_dst, size_t s_dst_pitch,
	const float *g_src, size_t s_src_pitch,
	unsigned int img_width, unsigned int img_height)
{
	extern __shared__ float s_mem[];
	unsigned int x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	unsigned int y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
	const unsigned int src_offset = IMUL(y, s_src_pitch) + x;
	unsigned int smem_offset = IMUL(threadIdx.y, blockDim.x) + threadIdx.x
		+ threadIdx.y;

	// Load data into shared memory
	if (y < img_height)
	{
		s_mem[smem_offset] = g_src[src_offset];
	}

	__syncthreads();

	// Compute smem_offset so that we read the values transposed
	smem_offset = IMUL(threadIdx.x, blockDim.x) + threadIdx.y + threadIdx.x;

	// Compute destination offset
	x = IMUL(blockIdx.y, blockDim.x) + threadIdx.x;
	y = IMUL(blockIdx.x, blockDim.y) + threadIdx.y;
	const unsigned int dst_offset = IMUL(y, s_dst_pitch) + x;

	// Write data back to global memory
	if (y < img_width)
	{
		g_dst[dst_offset] = s_mem[smem_offset];
	}
}

#include <stdio.h>
extern "C" void
cudaYCbYCrToRGBA( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    
    YCbYCrToRGBA<<<grid, threads>>>( d_dst, d_src, imageW/2, imageH );
    hipDeviceSynchronize();
}

extern "C" void
cudaGray1ToRGBA( uchar4 *d_dst, unsigned char *d_src, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    Gray1ToRGBA<<<grid, threads>>>( d_dst, d_src, imageW, imageH );
    hipDeviceSynchronize();
}



extern "C"
void cudaWarpImage( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH, float matrix[9] )
{
    size_t offset;

	tex.filterMode = hipFilterModePoint; // We don't use interpolation (intepo impossible with uchar)
	tex.normalized = false; // Don't normalize texture coordinates
	/* Clamping saves us some boundary checks */
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.addressMode[2] = hipAddressModeClamp;

    // Bind texture reference to linear memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    hipBindTexture2D( &offset, tex, (uchar4*)d_src, channelDesc, imageW, imageH, imageW*4*sizeof(unsigned char) );
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    warpImage<<<grid, threads>>>( d_dst, imageW, imageH, matrix );

    hipDeviceSynchronize();
    hipUnbindTexture( tex );
}

extern "C"
void cudaDiffRGB( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    diffRGB<<<grid, threads>>>( d_dst, d_src1, d_src2, imageW, imageH );
    hipDeviceSynchronize();
}

extern "C"
void cudaDiffFromYCbYCr( uchar4 *d_dst, uchar4 *d_srcA, uchar4 *d_srcB, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    fromDiff<<<grid, threads>>>( d_dst, d_srcA, d_srcB, imageW/2, imageH );
    hipDeviceSynchronize();
}

extern "C"
void cudaYCbYCrToY( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    YCbYCrToY<<<grid, threads>>>(d_dst, d_src, imageW/2, imageH);
    hipDeviceSynchronize();
}

extern "C"
void cudaYToYCbYCr( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    YToYCbYCr<<<grid, threads>>>(d_dst, d_src, imageW/2, imageH);
    hipDeviceSynchronize();
}

extern "C"
void cudaRGBAToFloat( float*outDevicePtr, uchar4*inDevicePtr, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    RGBAtoFloat<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH,imageW);
    hipDeviceSynchronize();
}

extern "C"
void cudaFloatToRGBA( uchar4*outDevicePtr, float*inDevicePtr, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    FloatToRGBA<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH,imageW);
    hipDeviceSynchronize();
}

extern "C"
void
cudaTranspose(
    float *d_dst, size_t dst_pitch,
	float *d_src, size_t src_pitch,
	unsigned int width, unsigned int height )
{
	// execution configuration parameters
	dim3 threads(16, 16);
	dim3 grid(iDivUp(width, 16), iDivUp(height, 16));
	size_t shared_mem_size =
		(threads.x * threads.y + (threads.y - 1)) * sizeof(float);

	transpose<<<grid, threads, shared_mem_size>>>(
		d_dst, dst_pitch / sizeof(float),
		d_src, src_pitch / sizeof(float),
		width, height);
    hipDeviceSynchronize();

}

extern "C"
void
cudaRGBAtoCuda( float *outDevicePtr, uchar4 *inDevicePtr, unsigned int imageW, unsigned int imageH, unsigned int pitch )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    RGBAtoFloat<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH, pitch);
    hipDeviceSynchronize();
}


extern "C"
void
cudaCudatoRGBA( uchar4 *outDevicePtr, float *inDevicePtr, unsigned int imageW, unsigned int imageH, unsigned int pitch )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    FloatToRGBA<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH, pitch);
    hipDeviceSynchronize();
}


extern "C"
void
cudaIntegrate( float *out, float *in, unsigned int width, unsigned int height, unsigned int pitch )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(width, BLOCKDIM_X), iDivUp(height, BLOCKDIM_Y));
    Integrate<<<grid, threads>>>( out, in, width, height, pitch );
    hipDeviceSynchronize();
}
