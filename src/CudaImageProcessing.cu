#include "hip/hip_runtime.h"

#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8


#include "cutil_math.h"

texture<uchar4, 2, hipReadModeElementType> tex;

int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// 24-bit multiplication is faster on G80,
// but we must be sure to multiply integers
// only within [-8M, 8M - 1] range
#if defined(CUDA_24BIT_IMUL)
#	define IMUL(a, b) __mul24(a, b)
#else
#	define IMUL(a, b) (a)*(b)
#endif

// TODO deformation
// At the moment this function is only used for testing
__global__
void cuWarpImage( uchar4 *d_dst, int imageW, int imageH, double matrix[9] )
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if( ix < imageW && iy < imageH )
    {
        // Warp is only used for TESTING purposes
        unsigned int pos = ix + iy*imageW;
        const float fx = float(ix);
        const float fy = float(iy);

        const float tu = fx*float(matrix[0]) +fy*float(matrix[1]) + float(matrix[2]);
        const float tv = fx*float(matrix[3]) +fy*float(matrix[4]) + float(matrix[5]);
        const float tw = fx*float(matrix[6]) +fy*float(matrix[7]) + float(matrix[8]); 
        
        const float u = tu/tw;
        const float v = tv/tw;
        
        const uchar4 col = tex2D( tex, u, v ); 
        const float R = (float)col.x;
        const float G = (float)col.y;
        const float B = (float)col.z;
        d_dst[ pos ].x = (unsigned char)( R > 0 ) ? ( ( R <=255 ) ? R : 255 ): 0 ;
        d_dst[ pos ].y = (unsigned char)( G > 0 ) ? ( ( G <=255 ) ? G : 255 ): 0 ;
        d_dst[ pos ].z = (unsigned char)( B > 0 ) ? ( ( B <=255 ) ? B : 255 ): 0 ;
    }
}

__global__
void anaglyphRGB( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        unsigned int pos = ix + iy*imageW;

        d_dst[ pos ].x = d_src1[pos].x; 
        d_dst[ pos ].y = d_src2[pos].y;
        d_dst[ pos ].z = d_src2[pos].z;
    }
}

__global__
void mixRGB( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        unsigned int posPix1 = ix + iy*imageW;

        const float3 src1 =  make_float3( d_src1[posPix1].x, d_src1[posPix1].y, d_src1[posPix1].z );
        const float3 src2 =  make_float3( d_src2[posPix1].x, d_src2[posPix1].y, d_src2[posPix1].z );

        const float3 result = src1+src2;

        const float R = (result.x/2.f);
        const float G = (result.y/2.f);
        const float B = (result.z/2.f);

        d_dst[ posPix1 ].x = (unsigned char)( R > 0 ) ? ( ( R <=255 ) ? R : 255 ): 0 ;
        d_dst[ posPix1 ].y = (unsigned char)( G > 0 ) ? ( ( G <=255 ) ? G : 255 ): 0 ;
        d_dst[ posPix1 ].z = (unsigned char)( B > 0 ) ? ( ( B <=255 ) ? B : 255 ): 0 ;
    }
}

__global__
void diffRGB( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        unsigned int posPix1 = ix + iy*imageW;

        const float3 src1 =  make_float3( d_src1[posPix1].x, d_src1[posPix1].y, d_src1[posPix1].z );
        const float3 src2 =  make_float3( d_src2[posPix1].x, d_src2[posPix1].y, d_src2[posPix1].z );

        const float3 result = fabs(src1-src2);

        const float R = (result.x+0.f);
        const float G = (result.y+0.f);
        const float B = (result.z+0.f);

        d_dst[ posPix1 ].x = (unsigned char)( R > 0 ) ? ( ( R <=255 ) ? R : 255 ): 0 ;
        d_dst[ posPix1 ].y = (unsigned char)( G > 0 ) ? ( ( G <=255 ) ? G : 255 ): 0 ;
        d_dst[ posPix1 ].z = (unsigned char)( B > 0 ) ? ( ( B <=255 ) ? B : 255 ): 0 ;
    }
}

#include <stdio.h>
// Convert 422 ycbycr to rgb
__global__
void
YCbYCrToRGBA(
    uchar4 *dst,
    uchar4 *src,
    int imageW, //960
    int imageH  //1080
)
{
    // Position in src image (960,1080)
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const float Y1 = (float)src[imageW * iy + ix].x;
        const float Cb = (float)src[imageW * iy + ix].y;
        const float Y2 = (float)src[imageW * iy + ix].z;
        const float Cr = (float)src[imageW * iy + ix].w;

        // Conversion in RGB
        const float R1 = Y1 + 1.371*(Cr-128);
        const float G1 = Y1 - 0.698*(Cr-128) - 0.336*(Cb - 128);
        const float B1 = Y1 + 1.732*(Cb-128);

        const float R2 = Y2 + 1.371*(Cr-128);
        const float G2 = Y2 - 0.698*(Cr-128) - 0.336*(Cb - 128);
        const float B2 = Y2 + 1.732*(Cb-128);

        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        dst[ posPix1 ].x = (unsigned char)( R1 > 0 ) ? ( ( R1 <=255 ) ? R1 : 255 ): 0 ;
        dst[ posPix1 ].y = (unsigned char)( G1 > 0 ) ? ( ( G1 <=255 ) ? G1 : 255 ): 0 ;
        dst[ posPix1 ].z = (unsigned char)( B1 > 0 ) ? ( ( B1 <=255 ) ? B1 : 255 ): 0 ;
        dst[ posPix1 ].w = 0;

        dst[ posPix2 ].x = (unsigned char)( R2 > 0 ) ? ( ( R2 <=255 ) ? R2 : 255 ): 0 ;
        dst[ posPix2 ].y = (unsigned char)( G2 > 0 ) ? ( ( G2 <=255 ) ? G2 : 255 ): 0 ;
        dst[ posPix2 ].z = (unsigned char)( B2 > 0 ) ? ( ( B2 <=255 ) ? B2 : 255 ): 0 ;
        dst[ posPix2 ].w = 0;
    }
}

__global__
void
YCbYCrToY(
    uchar4 *dst,
    uchar4 *src,
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const unsigned char Y1 = src[imageW * iy + ix].x;
        const unsigned char Y2 = src[imageW * iy + ix].z;

        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        dst[ posPix1 ].x = Y1;
        dst[ posPix1 ].y = Y1;
        dst[ posPix1 ].z = Y1;
        dst[ posPix1 ].w = 0;

        dst[ posPix2 ].x = Y2;
        dst[ posPix2 ].y = Y2;
        dst[ posPix2 ].z = Y2;
        dst[ posPix2 ].w = 0;
    }
}


__global__
void
YToYCbYCr(
    uchar4 *dst,
    uchar4 *src,
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        const unsigned char Y1 = src[ posPix1 ].x;
        const unsigned char Y2 = src[ posPix2 ].x;

        dst[imageW * iy + ix].x = Y1;
        dst[imageW * iy + ix].y = 0; // Cb
        dst[imageW * iy + ix].z = Y2;
        dst[imageW * iy + ix].w = 0; // Cr
    }
}

__global__
void
Gray1ToRGBA(
    uchar4 *dst,            // RGB
    unsigned char *src,     // Gray
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const unsigned int position = imageW * iy + ix;
        const unsigned char color = src[position];

        dst[ position ].x = color;
        dst[ position ].y = color;
        dst[ position ].z = color;
        dst[ position ].w = 0;
    }
}

__global__
void
fromDiff(
    uchar4 *dst,
    uchar4 *d_srcA,
    uchar4 *d_srcB,
    int imageW,
    int imageH
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH)
    {
        const float Y1A = (float)d_srcA[imageW * iy + ix].x;
        const float CbA = (float)d_srcA[imageW * iy + ix].y;
        const float Y2A = (float)d_srcA[imageW * iy + ix].z;
        const float CrA = (float)d_srcA[imageW * iy + ix].w;

        // Conversion in RGB
        const float R1A = Y1A + 1.371*(CrA-128);
        const float G1A = Y1A - 0.698*(CrA-128) - 0.336*(CbA - 128);
        const float B1A = Y1A + 1.732*(CbA-128);

        const float R2A = Y2A + 1.371*(CrA-128);
        const float G2A = Y2A - 0.698*(CrA-128) - 0.336*(CbA - 128);
        const float B2A = Y2A + 1.732*(CbA-128);

        // SRC B
        const float Y1B = (float)d_srcB[imageW * iy + ix].x;
        const float CbB = (float)d_srcB[imageW * iy + ix].y;
        const float Y2B = (float)d_srcB[imageW * iy + ix].z;
        const float CrB = (float)d_srcB[imageW * iy + ix].w;

        // Conversion in RGB
        const float R1B = Y1B + 1.371*(CrB-128);
        const float G1B = Y1B - 0.698*(CrB-128) - 0.336*(CbB - 128);
        const float B1B = Y1B + 1.732*(CbB-128);

        const float R2B = Y2B + 1.371*(CrB-128);
        const float G2B = Y2B - 0.698*(CrB-128) - 0.336*(CbB - 128);
        const float B2B = Y2B + 1.732*(CbB-128);

        const unsigned int posPix1 = 2*imageW * iy + 2*ix;
        const unsigned int posPix2 = posPix1 + 1;

        const float R1 = (R1B-R1A)*0.5 + 127.f;
        const float G1 = (G1B-G1A)*0.5 + 127.f;
        const float B1 = (B1B-B1A)*0.5 + 127.f;
        const float R2 = (R2B-R2A)*0.5 + 127.f;
        const float G2 = (G2B-G2A)*0.5 + 127.f;
        const float B2 = (B2B-B2A)*0.5 + 127.f;

        dst[ posPix1 ].x = (unsigned char)( R1 > 0 ) ? ( ( R1 <=255 ) ? R1 : 255 ): 0 ;
        dst[ posPix1 ].y = (unsigned char)( G1 > 0 ) ? ( ( G1 <=255 ) ? G1 : 255 ): 0 ;
        dst[ posPix1 ].z = (unsigned char)( B1 > 0 ) ? ( ( B1 <=255 ) ? B1 : 255 ): 0 ;

        dst[ posPix2 ].x = (unsigned char)( R2 > 0 ) ? ( ( R2 <=255 ) ? R2 : 255 ): 0 ;
        dst[ posPix2 ].y = (unsigned char)( G2 > 0 ) ? ( ( G2 <=255 ) ? G2 : 255 ): 0 ;
        dst[ posPix2 ].z = (unsigned char)( B2 > 0 ) ? ( ( B2 <=255 ) ? B2 : 255 ): 0 ;
    }
}

__global__
void
RGBAtoFloat(
    float *dst,
    uchar4 *d_src,
    int imageW,
    int imageH,
    int pitch
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH )
    {
        const unsigned int position = imageW * iy + ix;

        const float col = d_src[position].x;
        dst[position] = col/255.0;
    }
}

__global__
void
FloatToRGBA(
    uchar4 *dst,
    float *d_src,
    int imageW,
    int imageH,
    int pitch
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Position in src image
    if(ix < imageW && iy < imageH )
    {
        const unsigned int position = pitch * iy + ix;

        const float pixVal = d_src[position]*255.0;
        const unsigned char p = (unsigned char)( pixVal > 0 ) ? ( ( pixVal <=255 ) ? pixVal : 255 ): 0 ;

        dst[position].x = p;
        dst[position].y = p;
        dst[position].z = p;
        dst[position].w = 0;
    }
}


// Testing purpose
__global__
void
Integrate(
    float *dst,
    float *d_src,
    int imageW,
    int imageH,
    int pitch
)
{
    // Position in dest image
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    if(ix < imageW && iy < imageH )
    {
        const unsigned int position = pitch * iy + ix;

        float sum = 0;

        for( int i=0; i < ix; i++ )
            sum += d_src[i];
        
        dst[position] = sum;
    }
}


// Transpose of matrix
__global__ void
transpose(
	float *g_dst, size_t s_dst_pitch,
	const float *g_src, size_t s_src_pitch,
	unsigned int img_width, unsigned int img_height)
{
	extern __shared__ float s_mem[];
	unsigned int x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	unsigned int y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
	const unsigned int src_offset = IMUL(y, s_src_pitch) + x;
	unsigned int smem_offset = IMUL(threadIdx.y, blockDim.x) + threadIdx.x
		+ threadIdx.y;

	// Load data into shared memory
	if (y < img_height)
	{
		s_mem[smem_offset] = g_src[src_offset];
	}

	__syncthreads();

	// Compute smem_offset so that we read the values transposed
	smem_offset = IMUL(threadIdx.x, blockDim.x) + threadIdx.y + threadIdx.x;

	// Compute destination offset
	x = IMUL(blockIdx.y, blockDim.x) + threadIdx.x;
	y = IMUL(blockIdx.x, blockDim.y) + threadIdx.y;
	const unsigned int dst_offset = IMUL(y, s_dst_pitch) + x;

	// Write data back to global memory
	if (y < img_width)
	{
		g_dst[dst_offset] = s_mem[smem_offset];
	}
}

#include <stdio.h>
extern "C" void
cudaYCbYCrToRGBA( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    
    YCbYCrToRGBA<<<grid, threads>>>( d_dst, d_src, imageW/2, imageH );
    hipDeviceSynchronize();
}

extern "C" void
cudaGray1ToRGBA( uchar4 *d_dst, unsigned char *d_src, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    Gray1ToRGBA<<<grid, threads>>>( d_dst, d_src, imageW, imageH );
    hipDeviceSynchronize();
}



extern "C"
void cudaWarpImage( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH, double matrix[9] )
{
    size_t offset=0;

	tex.filterMode = hipFilterModePoint; // We don't use interpolation (interpo impossible with uchar)
	tex.normalized = false; // Don't normalize texture coordinates
	/* Clamping saves us some boundary checks */
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.addressMode[2] = hipAddressModeClamp;

    // Bind texture reference to linear memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    hipBindTexture2D( &offset, tex, (uchar4*)d_src, channelDesc, imageW, imageH, imageW*4*sizeof(unsigned char) );
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    //matrix[0] = 1;
    //matrix[1] = 0;
    //matrix[2] = 0;
    //matrix[3] = 0;
    //matrix[4] = 1;
    //matrix[5] = 0;
    //matrix[6] = 0;
    //matrix[7] = 0;
    //matrix[8] = 1;

    double *d_matrix; // device matrix
    hipMalloc((void**)&d_matrix, sizeof(double)*9);
    hipMemcpy(d_matrix, matrix, sizeof(double)*9, hipMemcpyHostToDevice);
    cuWarpImage<<<grid, threads>>>( d_dst, imageW, imageH, d_matrix );
    hipFree(d_matrix);
    hipDeviceSynchronize();
    hipUnbindTexture( tex );
}

extern "C"
void cudaDiffRGB( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    diffRGB<<<grid, threads>>>( d_dst, d_src1, d_src2, imageW, imageH );
    hipDeviceSynchronize();
}

extern "C"
void cudaMix( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    mixRGB<<<grid, threads>>>( d_dst, d_src1, d_src2, imageW, imageH );
    hipDeviceSynchronize();
}

extern "C"
void cudaAnaglyph( uchar4 *d_dst, uchar4 *d_src1, uchar4 *d_src2, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    anaglyphRGB<<<grid, threads>>>( d_dst, d_src1, d_src2, imageW, imageH );
    hipDeviceSynchronize();
}

extern "C"
void cudaDiffFromYCbYCr( uchar4 *d_dst, uchar4 *d_srcA, uchar4 *d_srcB, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    fromDiff<<<grid, threads>>>( d_dst, d_srcA, d_srcB, imageW/2, imageH );
    hipDeviceSynchronize();
}

extern "C"
void cudaYCbYCrToY( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    YCbYCrToY<<<grid, threads>>>(d_dst, d_src, imageW/2, imageH);
    hipDeviceSynchronize();
}

extern "C"
void cudaYToYCbYCr( uchar4 *d_dst, uchar4 *d_src, int imageW, int imageH )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW/2, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    YToYCbYCr<<<grid, threads>>>(d_dst, d_src, imageW/2, imageH);
    hipDeviceSynchronize();
}

extern "C"
void cudaRGBAToFloat( float*outDevicePtr, uchar4*inDevicePtr, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    RGBAtoFloat<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH,imageW);
    hipDeviceSynchronize();
}

extern "C"
void cudaFloatToRGBA( uchar4*outDevicePtr, float*inDevicePtr, int imageW, int imageH)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    FloatToRGBA<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH,imageW);
    hipDeviceSynchronize();
}

extern "C"
void
cudaTranspose(
    float *d_dst, size_t dst_pitch,
	float *d_src, size_t src_pitch,
	unsigned int width, unsigned int height )
{
	// execution configuration parameters
	dim3 threads(16, 16);
	dim3 grid(iDivUp(width, 16), iDivUp(height, 16));
	size_t shared_mem_size =
		(threads.x * threads.y + (threads.y - 1)) * sizeof(float);

	transpose<<<grid, threads, shared_mem_size>>>(
		d_dst, dst_pitch / sizeof(float),
		d_src, src_pitch / sizeof(float),
		width, height);
    hipDeviceSynchronize();

}

extern "C"
void
cudaRGBAtoCuda( float *outDevicePtr, uchar4 *inDevicePtr, unsigned int imageW, unsigned int imageH, unsigned int pitch )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    RGBAtoFloat<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH, pitch);
    hipDeviceSynchronize();
}


extern "C"
void
cudaCudatoRGBA( uchar4 *outDevicePtr, float *inDevicePtr, unsigned int imageW, unsigned int imageH, unsigned int pitch )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));
    FloatToRGBA<<<grid, threads>>>(outDevicePtr,inDevicePtr,imageW,imageH, pitch);
    hipDeviceSynchronize();
}


extern "C"
void
cudaIntegrate( float *out, float *in, unsigned int width, unsigned int height, unsigned int pitch )
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(width, BLOCKDIM_X), iDivUp(height, BLOCKDIM_Y));
    Integrate<<<grid, threads>>>( out, in, width, height, pitch );
    hipDeviceSynchronize();
}
